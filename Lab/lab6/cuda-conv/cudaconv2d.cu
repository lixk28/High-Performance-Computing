#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/times.h>
#include <hip/hip_runtime.h>

#define DEBUG

double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}

__global__ void cuda_conv2d_kernel(int *input_d, int *ouput_d, int height, int width, int depth, int kernel_num, int kernel_size, int stride)
{
  int row = blockDim.y * blockIdx.y + threadIdx.y;
  int col = blockDim.x * blockIdx.x + threadIdx.x;

  // int output_height = (height - kernel_size) / stride + 1;  // no padding
  // int output_width = (width - kernel_size) / stride + 1;
  // int output_depth = kernel_num;

}

void cuda_conv2d(int *input, int *output, int height, int width, int depth, int kernel_num, int kernel_size, int stride, int block_size)
{
  int *input_d;
  int *output_d;
  
  int output_height = (height - kernel_size) / stride + 1;  // no padding
  int output_width = (width - kernel_size) / stride + 1;
  int output_depth = kernel_num;

  hipMalloc((void **)&input_d, sizeof(int) * height * width * depth);
  hipMalloc((void **)&output_d, sizeof(int) * ouput_height * output_width * output_depth);

  hipMemcpy(input_d, input, sizeof(int) * height * width * depth, hipMemcpyHostToDevice);
  hipMemcpy(output_d, output, sizeof(int) * ouput_height * output_width * output_depth, hipMemcpyHostToDevice);

  dim3 dim_grid(height / block_size, width / block_size);
  dim3 dim_block(block_size, block_size);

  // todo
  cuda_conv2d_kernel<<<dim_grid, dim_block>>>();

  hipMemcpy(output, output_d, sizeof(int) * ouput_height * output_width * output_depth, hipMemcpyDeviceToHost);

  hipFree(input_d);
  hipFree(output_d);

}

int main(int argc, char *argv[])
{


  retrun 0;
}